#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "spmv.h"
#define THREADS_PER_BLOCK 256

template <class T>
__global__ void
spmv_kernel_ell(unsigned int* col_ind, T* vals, int m, int n, int nnz, 
                double* x, double* b)
{
  
  //shared memory for the reduction
  __shared__ double reduce[THREADS_PER_BLOCK];
 
  int row = blockIdx.x;
  int tid = threadIdx.x;
  
  
  if (tid == 0 && row == 0)
    printf("n %d\n\n\n\n\n\n\n", n);
  
  if (row < m)
    {

      int begin = row * n;
      int chunk = n/THREADS_PER_BLOCK;
      
     
      // have each tread sum chunk number of entries in matrix and store at there
      // index. There will be a remainder if THREADS_PER_BLOCK does not divide
      // n evenlly...just have the final thread do the remainder in serial.
      if (chunk == 0)
	{

	  if (tid < n)
	    reduce[tid] = vals[begin + tid] * x[col_ind[begin + tid]];
	  
	}
      
      // bug somewhere here
      else if (chunk == 1)
	{
	  if (tid < n)
	    reduce[tid] = vals[begin + tid] * x[col_ind[begin + tid]];

	  if (tid == THREADS_PER_BLOCK - 1)
	    {
	      for (int i = begin + tid + 1 ; i < begin + n ; i++)
		{
		  reduce[tid] = vals[i] * x[col_ind[i]];
		}
	    }
	}
      else
	{
	  reduce[tid] = 0;
	  int i;
	  for (i = begin + tid*chunk ; i < begin + (tid+1)*chunk ; i++)
	    {
	      reduce[tid] += vals[i] * x[col_ind[i]];
	    }
	  if (tid == (THREADS_PER_BLOCK - 1))
	    {
	      //printf("i is on row %d: %d %d\n", row ,i, n*(row+1));
	      while (i < n*(row+1))
		{
		  reduce[tid] += vals[i] * x[col_ind[i]];
		  i++;
		}
	    }
	  
	}
      
         
      // do sequential reduction (assuming number of threads is a power of 2
      for (int s = THREADS_PER_BLOCK/2 ; s > 0 ; s>>=1)
	{	     
	  if (tid < s)
	    reduce[tid] += reduce[tid + s];
	  __syncthreads();
	}
      
      if (tid == 0)
	b[row] = reduce[tid];
    }
  
  /*
  if (tid == 0 && row == 0)
    {
      printf("\n");
      for (int i = 0 ; i < m ; i++)
	printf("\t%f", b[i]);
      printf("\n");
    }
  */
  

}




void spmv_gpu_ell(unsigned int* col_ind, double* vals, int m, int n, int nnz, 
                  double* x, double* b)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    // GPU execution parameters
    unsigned int blocks = m; 
    unsigned int threads = THREADS_PER_BLOCK; 
    unsigned int shared = threads * sizeof(double);

    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);

    checkCudaErrors(hipEventRecord(start, 0));
    for(unsigned int i = 0; i < MAX_ITER; i++) {
        hipDeviceSynchronize();
        spmv_kernel_ell<double><<<dimGrid, dimBlock, shared>>>(col_ind, vals, 
                                                               m, n, nnz, x, b);

    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Exec time (per itr): %0.8f s\n", (elapsedTime / 1e3 / MAX_ITER));

}




void allocate_ell_gpu(unsigned int* col_ind, double* vals, int m, int n, int n_new, 
                      int nnz, double* x, unsigned int** dev_col_ind, 
                      double** dev_vals, double** dev_x, double** dev_b)
{
  
  CopyData(col_ind, m*n_new, sizeof(int), dev_col_ind);
  CopyData(vals, m*n_new, sizeof(double), dev_vals);
  CopyData(x, n, sizeof(double), dev_x);
  double* b = (double*) malloc(sizeof(double) * m);
  CopyData(b, m, sizeof(double), dev_b);
  free(b);
  
  

}

void allocate_csr_gpu(unsigned int* row_ptr, unsigned int* col_ind, 
                      double* vals, int m, int n, int nnz, double* x, 
                      unsigned int** dev_row_ptr, unsigned int** dev_col_ind,
                      double** dev_vals, double** dev_x, double** dev_b)
{

  CopyData(row_ptr, m+1, sizeof(int), dev_row_ptr);
  CopyData(col_ind, nnz, sizeof(int), dev_col_ind);
  CopyData(vals, nnz, sizeof(double), dev_vals);
  CopyData(x, n, sizeof(double), dev_x);
  double* b = (double*) malloc(sizeof(double) * m);
  CopyData(b, m, sizeof(double), dev_b);
  free(b);
  
    
}

void get_result_gpu(double* dev_b, double* b, int m)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;


    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMemcpy(b, dev_b, sizeof(double) * m, 
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Pinned Host to Device bandwidth (GB/s): %f\n",
         (m * sizeof(double)) * 1e-6 / elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

template <class T>
void CopyData(
  T* input,
  unsigned int N,
  unsigned int dsize,
  T** d_in)
{
  // timers
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;

  // Allocate pinned memory on host (for faster HtoD copy)
  T* h_in_pinned = NULL;
  checkCudaErrors(hipHostMalloc((void**) &h_in_pinned, N * dsize));
  assert(h_in_pinned);
  memcpy(h_in_pinned, input, N * dsize);

  // copy data
  checkCudaErrors(hipMalloc((void**) d_in, N * dsize));
  checkCudaErrors(hipEventRecord(start, 0));
  checkCudaErrors(hipMemcpy(*d_in, h_in_pinned,
                             N * dsize, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("  Pinned Device to Host bandwidth (GB/s): %f\n",
         (N * dsize) * 1e-6 / elapsedTime);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}


template <class T>
__global__ void
spmv_kernel(unsigned int* row_ptr, unsigned int* col_ind, T* vals, 
            int m, int n, int nnz, double* x, double* b)
{

 
  // each block will do a reduction between a single row and the column vector x.
  int row = blockIdx.x;
  int tid = threadIdx.x;
  
  /*
  if (row == 0 && tid == 0)
    {
      printf("\n");
      for (int i = 0 ; i < m+1 ; i++)
	printf("\t%d", row_ptr[i]);
    }
  */

  //shared memory for the reduction
  __shared__ double reduce[THREADS_PER_BLOCK];

  // guard against block reading past rows of matrix
  // (unecessary if kernal is launched with m blocks)
  if (row < m)
    {
      
      // set reduction array to zero
      reduce[tid] = 0;
      
   

      int begin = row_ptr[row];
      int end = row_ptr[row+1];
      int length = end - begin;

     
      /*
      if (tid == 0)
	printf("row %d length: %d\n", row, length);
      */
      //fill reduction array with products
      // since reduction array is only length THREADS_PER_BLOCK
      // When we first put products in the array they must be summed down to fit inside.
      // This can be done by taking chunks of the product vector and having one
      // thread sum them all
      // these are stored in the thread id index
      // Once this is done a sequential reduction can be done.
      reduce[tid] = 0;
      int chunk = length/THREADS_PER_BLOCK;
      //if(row == 5 && tid == 0)
      //	printf("chunk is %d %d\n", chunk, length);
      // if we can fit everything in row directly into shared.
      if (chunk == 0)
	{
	  //printf("in 0 chunk\n");
	  if (tid < length)
	    reduce[tid] = vals[begin + tid] * x[col_ind[begin + tid]];
	}
      // if we need to reduce array to get into shared.
      else
	{
	  //printf("in multichunk\n");
	  int i;
	  for (i = begin + tid*chunk ; i < begin + (tid+1)*chunk ; i++)
	    {
	      reduce[tid] += vals[i] * x[col_ind[i]];
	    }
	  
	  // if we have a remainder just use the last thread to get the rest
	  if (tid == (THREADS_PER_BLOCK - 1))
	    {
	      while (i < begin + length)
		{
		  reduce[tid] += vals[i] * x[col_ind[i]];
		  i++;
		}
	    }
	}


      //if (row == 0)
      //	printf("thread %d has %f\n", tid, reduce[tid]);

         
      
      // if reduce is odd number length, and is last thread in block
      if (THREADS_PER_BLOCK % 2 == 1 && tid == THREADS_PER_BLOCK-2)
	reduce[tid] += reduce[tid + 1];
      __syncthreads();

      // do sequential reduction
      for (int s = THREADS_PER_BLOCK/2 ; s > 0 ; s>>=1)
	{
	  if (tid < s)
	    reduce[tid] += reduce[tid + s];
	  __syncthreads();
	}
      
      if (tid == 0)
	b[row] = reduce[tid];
    }
  
  /*
  if (tid == 0 && row == 0)
    {
      printf("\n");
      for (int i = 0 ; i < m ; i++)
	printf("\t%f", b[i]);
      printf("\n");
    }
  */


}



void spmv_gpu(unsigned int* row_ptr, unsigned int* col_ind, double* vals,
              int m, int n, int nnz, double* x, double* b)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    // GPU execution parameters
    // 1 thread block per row
    // 64 threads working on the non-zeros on the same row
    unsigned int blocks = m; 
    //printf("\n\nnumber of blocks: %d\n\n", m);
    unsigned int threads = THREADS_PER_BLOCK; 
    unsigned int shared = threads * sizeof(double);

    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);

    checkCudaErrors(hipEventRecord(start, 0));
    for(unsigned int i = 0; i < MAX_ITER; i++) {
      hipDeviceSynchronize();
      spmv_kernel<double><<<dimGrid, dimBlock, shared>>>(row_ptr, col_ind, 
							 vals, m, n, nnz, 
							 x, b);
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Exec time (per itr): %0.8f s\n", (elapsedTime / 1e3 / MAX_ITER));

}
