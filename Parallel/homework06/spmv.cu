#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "spmv.h"



// ----------------------------------------------------------------- 
// For creating shared memory
template<class T>
struct SharedMemory
{
    __device__ inline operator T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};
// ----------------------------------------------------------------- 


// ----------------------------------------------------------------- 
// ELLPACK SPMV
template <class T>
__global__ void
spmv_kernel_ell(unsigned int* col_ind, T* vals, int m, int n, int nnz, 
                double* x, double* b)
{
    // EXTRA CREDIT
}

void spmv_gpu_ell(unsigned int* col_ind, double* vals, int m, int n, int nnz, 
                  double* x, double* b)
{
    // GPU execution parameters
    unsigned int blocks = m; 
    unsigned int threads = 64; 
    unsigned int shared = threads * sizeof(double);

    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);

    spmv_kernel_ell<double><<<dimGrid, dimBlock, shared>>>(col_ind, vals, m, n, 
                                                           nnz, x, b);
}


void allocate_ell_gpu(unsigned int* col_ind, double* vals, int m, int n, 
                      int nnz, double* x, unsigned int** dev_col_ind, 
                      double** dev_vals)
{
    // copy ELL data to GPU and allocate memory for output
    CopyData<unsigned int>(col_ind, m * n, sizeof(unsigned int), dev_col_ind);
    CopyData<double>(vals, m * n, sizeof(double), dev_vals);
}
// ----------------------------------------------------------------- 


// ----------------------------------------------------------------- 
// CSR SPMV
template <class T>
__global__ void
spmv_kernel(unsigned int* row_ptr, unsigned int* col_ind, T* vals, 
              int m, int n, int nnz, double* x, double* b)
{
    // find the start and end indicies for the target row
    unsigned int row_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(row_id < m) {
        unsigned int start = row_ptr[row_id];
        unsigned int end = row_ptr[row_id + 1];

        //  each thread calculates over non-zero element(s)
        T accum = 0.0;
        for(unsigned int i = start; i < end; i++) {
            accum += vals[i] * x[col_ind[i]];
        }
        // __syncthreads();
        b[row_id] = accum;
    } 
}


void spmv_gpu(unsigned int* row_ptr, unsigned int* col_ind, double* vals,
                int m, int n, int nnz, double* x, double* b)
{
    unsigned int threads = 1024; 
    unsigned int blocks = (m + threads - 1) / threads; 
    unsigned int shared = 0;

    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);

    spmv_kernel<double><<<dimGrid, dimBlock, shared>>>(row_ptr, col_ind, vals,
                                                          m, n, nnz, x, b);
}


void allocate_csr_gpu(unsigned int* row_ptr, unsigned int* col_ind, 
                      double* vals, int m, int n, int nnz, double* x, 
                      unsigned int** dev_row_ptr, unsigned int** dev_col_ind,
                      double** dev_vals)
{
    // copy CSR data to GPU and allocate memory for output
    CopyData<unsigned int>(row_ptr, (m + 1), sizeof(unsigned int), dev_row_ptr);
    CopyData<unsigned int>(col_ind, nnz, sizeof(unsigned int), dev_col_ind);
    CopyData<double>(vals, nnz, sizeof(double), dev_vals);
}
// ----------------------------------------------------------------- 


// ----------------------------------------------------------------- 
void allocate_data_gpu(double* x, double* b, double** dx, double** db, 
                       double** drk, double** dpk, double** dap, double** z1,
                       double** z2, int m, int n)
{
    CopyData<double>(x, n, sizeof(double), dx);
    CopyData<double>(b, m, sizeof(double), db);
    checkCudaErrors(hipMalloc((void**) drk, sizeof(double) * m));
    checkCudaErrors(hipMalloc((void**) dpk, sizeof(double) * m));
    checkCudaErrors(hipMalloc((void**) dap, sizeof(double) * m));
    checkCudaErrors(hipMemset((void*) *drk, 0, sizeof(double) * m));
    checkCudaErrors(hipMemset((void*) *dpk, 0, sizeof(double) * m));
    checkCudaErrors(hipMemset((void*) *dap, 0, sizeof(double) * m));

    int next_p2 = n;
    if(!((n != 0) && ((n & (n - 1)) == 0))) {
       next_p2 = pow(2, (int) log2((double) n) + 1);
    }
    checkCudaErrors(hipMalloc((void**) z1, sizeof(double) * next_p2));
    checkCudaErrors(hipMalloc((void**) z2, sizeof(double) * next_p2));
    checkCudaErrors(hipMemset((void*) *z1, 0, sizeof(double) * next_p2));
    checkCudaErrors(hipMemset((void*) *z2, 0, sizeof(double) * next_p2));
}
// ----------------------------------------------------------------- 


// ----------------------------------------------------------------- 
void get_result_gpu(double* dev_b, double* b, int m)
{
    // timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;


    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMemcpy(b, dev_b, sizeof(double) * m, 
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("  Pinned Device to Host bandwidth (GB/s): %f\n",
         (m * sizeof(double)) * 1e-6 / elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
// ----------------------------------------------------------------- 


// ----------------------------------------------------------------- 
template <class T>
void CopyData(
  T* input,
  unsigned int N,
  unsigned int dsize,
  T** d_in)
{
  // timers
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;

  // Allocate pinned memory on host (for faster HtoD copy)
  T* h_in_pinned = NULL;
  checkCudaErrors(hipHostMalloc((void**) &h_in_pinned, N * dsize));
  assert(h_in_pinned);
  memcpy(h_in_pinned, input, N * dsize);

  // copy data
  checkCudaErrors(hipMalloc((void**) d_in, N * dsize));
  checkCudaErrors(hipEventRecord(start, 0));
  checkCudaErrors(hipMemcpy(*d_in, h_in_pinned,
                             N * dsize, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("  Pinned Device to Host bandwidth (GB/s): %f\n",
         (N * dsize) * 1e-6 / elapsedTime);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}
// ----------------------------------------------------------------- 



// ----------------------------------------------------------------- 
// Other GPU Kernels
template <class T>
__global__ void
vec_add_kernel(T c, T* x, T* y, T* z, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    z[i] = c * x[i] + y[i];
}


void vec_add_gpu(const int n, const double a, double* x, double* y, double* z)
{
    unsigned int threads = 1024;
    unsigned int blocks = (n + threads - 1) / threads;
    unsigned int shared = 0;
    dim3 dimGrid(blocks, 1, 1);
    dim3 dimBlock(threads, 1, 1);
   
    vec_add_kernel<double><<<dimGrid, dimBlock, shared>>>(a, x, y, z, n); 
}
// ----------------------------------------------------------------- 


// ----------------------------------------------------------------- 
// Kernels needed for dnrm2
template <class T>
__global__ void
reduce_kernel(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();
    int tid = threadIdx.x;
    int i = blockIdx.x*blockDim.x + tid;
    sdata[tid] = g_idata[i];
    __syncthreads();

    for (int s = blockDim.x / 2 ; s > 0 ; s >>= 1)
      {
	if (tid < s)
	  {
	    sdata[tid] += sdata[tid + s];
	  }
	__syncthreads();
      }

    

    if (tid == 0){
      g_odata[blockIdx.x] = sdata[0];
    }
}


template <class T>
__global__ void
vec_mul_kernel(T c, T* x, T* y, T* z, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    {
      z[i] = c * x[i] * y[i];
    }
}



double ddot_gpu(int n, double* x, double *y, double* z1, double* z2)
{
  
  // Create temporary buffer
  int next_p2 = n;
  if(!((n != 0) && ((n & (n - 1)) == 0))) {
    next_p2 = pow(2, (int) log2((double) n) + 1);
  } 
  checkCudaErrors(hipMemset((void*) z1, 0.0, sizeof(double) * next_p2));
  
  //fprintf(stderr, "length after multiplication %d\n", next_p2);
  
  // COMPLETE THIS 
  unsigned int threads = 1024;
  unsigned int blocks = (n + threads - 1) / threads;
  unsigned int shared = 0;
  vec_mul_kernel<double><<<blocks, threads, shared>>>(1.0, x, y, z1, n);
  
  //multiplication stored in z1 which is now a power of 2 and we reduce on down to num blocks/2
  blocks = (next_p2 + threads - 1) / threads;
 
  shared = threads * sizeof(double);
 
  //fprintf(stderr, "\n\non reduction 1\n");
  //fprintf(stderr, "number of blocks %d length %d \n", blocks, next_p2);
  checkCudaErrors(hipMemset((void*) z2, 0, sizeof(double) * next_p2));
  reduce_kernel<double><<<blocks, threads, shared>>>(z1, z2, next_p2);

  double* dot = (double*) malloc(sizeof(double));

  // keep reducing on new arrays of size number of block until there are no more blocks
  int old_length;
  int count = 0;
  while (blocks > 1)
    {
      //fprintf(stderr, "\n\non reduction %d\n", count + 2);
      old_length = blocks;
      blocks = (old_length+ threads - 1) / threads;
      count++;
      //fprintf(stderr, "number of blocks %d, length %d\n", blocks, old_length);
      // flip floping memory between z1 and z2
      if (count % 2 == 1)
	{
	  checkCudaErrors(hipMemset((void*) z1, 0.0, sizeof(double) * old_length));
	  reduce_kernel<double><<<blocks, threads, shared>>>(z2, z1, old_length);
	  checkCudaErrors(hipMemcpy(dot, z1, sizeof(double), 
				     hipMemcpyDeviceToHost));
	}
      else
	{
	  checkCudaErrors(hipMemset((void*) z2, 0.0, sizeof(double) * old_length));
	  reduce_kernel<double><<<blocks, threads, shared>>>(z1, z2, old_length);
	  checkCudaErrors(hipMemcpy(dot, z2, sizeof(double), 
				     hipMemcpyDeviceToHost));
	}
      

    }

  return *dot;

}


void test_dot(int n, double* b, double *b1, double** d_b, double** d_b1,
	      double** z1, double** z2)
{
  

  CopyData<double>(b, n, sizeof(double), d_b);
  CopyData<double>(b1, n, sizeof(double), d_b1);
  checkCudaErrors(hipMalloc((void**) z1, sizeof(double) * n));
  checkCudaErrors(hipMalloc((void**) z2, sizeof(double) * n));

  ddot_gpu(n, *d_b, *d_b1, *z1, *z2);
}



double dnrm2_gpu(const int n, double* x, double* z1, double* z2)
{
    double nrm = ddot_gpu(n, x, x, z1, z2);
    return sqrt(nrm);
}
// ----------------------------------------------------------------- 


// ----------------------------------------------------------------- 
int cg_gpu_csr(unsigned int* row_ptr, unsigned int* col_ind, double* vals, 
               double *x, double* b, double* rk, double* pk, double* ap, 
               double* z1, double* z2, int m, int n, int nnz, int max_iter, 
               double tol)
{
    // r0 = b - Ax
    spmv_gpu(row_ptr, col_ind, vals, m, n, nnz, x, ap);
    vec_add_gpu(m, -1.0, ap, b, rk);
    // if r0 is sufficiently small, return x0 as the result
    double residual = dnrm2_gpu(n, rk, z1, z2);
    if(residual < tol) {
        fprintf(stdout, "\tInput is the solution\n");
        return 0;
    } else {
      //fprintf(stdout, "\n\tInitial residual is %f\n", residual);
    }

    // p0 = r0
    checkCudaErrors(hipMemcpy(pk, rk, sizeof(double) * m, 
                               hipMemcpyDeviceToDevice));

    int k = 0;
    double residual_new = 0.0;
    // repeat until convergence of max iterations has been reached
    for(int i = 0; i < max_iter; i++) {
        // A * p
        spmv_gpu(row_ptr, col_ind, vals, m, n, nnz, pk, ap);
        // d = p^T * A * p
        double dotprod = ddot_gpu(m, pk, ap, z1, z2);
        // alpha = (r^t * r) / d;
        double alpha = (residual * residual) / dotprod;

        // xk = xk + alpha * pk
        vec_add_gpu(m, alpha, pk, x, x);
        // rk = rk - alpha * A*p
        vec_add_gpu(m, (-1.0 * alpha), ap, rk, rk);

        // r^t * r
        residual_new = dnrm2_gpu(m, rk, z1, z2);
        if(residual_new < tol) {
            fprintf(stdout, "\tSolution calculated. Final residual: %f\n", 
                    residual_new);
            break;
        } else {
	  //fprintf(stdout, "\tIt: %d\tresidual: %f\n", k, residual_new);
        }

        // beta = (r^t * r) / residual
        double beta = (residual_new * residual_new) / (residual * residual);

        // p = r + beta * p 
        vec_add_gpu(m, beta, pk, rk, pk);

        residual = residual_new;
        k++;
    }
    return 0;
}
// ----------------------------------------------------------------- 


// ----------------------------------------------------------------- 
// Free GPU memory
void free_gpu(unsigned int* drp, unsigned int* dci, unsigned int* dec, 
              double* dev, double* dx, double* db, double* drk, double* dpk, 
              double* dap, double* z1, double* z2)
{
    checkCudaErrors(hipFree(drp));
    checkCudaErrors(hipFree(dci));
    checkCudaErrors(hipFree(dec));
    checkCudaErrors(hipFree(dev));
    checkCudaErrors(hipFree(dx));
    checkCudaErrors(hipFree(db));
    checkCudaErrors(hipFree(drk));
    checkCudaErrors(hipFree(dpk));
    checkCudaErrors(hipFree(dap));
    checkCudaErrors(hipFree(z1));
    checkCudaErrors(hipFree(z2));
}
// ----------------------------------------------------------------- 

